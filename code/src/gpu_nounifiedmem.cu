
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

__global__ void saxpy(int n, float a, float* x, float* y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    int n = 500000000; // Smaller for testing
    float a = 2.0;

    std::vector<float> x(n, 1.0);
    std::vector<float> y(n, 1.0);

    float* x_dev, * y_dev;

    hipError_t cudaStatus;

    // Allocate the required memory on the GPU
    cudaStatus = hipMalloc(&x_dev, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for x_dev!" << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&y_dev, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for y_dev!" << std::endl;
        hipFree(x_dev);
        return 1;
    }

    // Transfer the data from the CPU to the GPU
    hipMemcpy(x_dev, x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_dev, y.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // How many threads there may be in a block
    int blockSize = 256;

    // How many blocks are necessary to cover all the datapoints
    int numBlocks = (n + blockSize - 1) / blockSize;

    std::cout << "for " << n << " elements, " << numBlocks << " blocks are needed" << std::endl;

    // Measure performance
    auto start = std::chrono::high_resolution_clock::now();

    // Launch the kernel
    saxpy<<<numBlocks, blockSize>>>(n, a, x_dev, y_dev);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "saxpy launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(x_dev);
        hipFree(y_dev);
        return 1;
    }

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // Copy the result back to the host (CPU)
    hipMemcpy(y.data(), y_dev, n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms\n";

    hipFree(x_dev);
    hipFree(y_dev);

    return 0;
}
